#include "hip/hip_runtime.h"
#include <stdio.h>
#include <../common/book.h>

__global__ void helloWorldFromGPU(void){
	printf("Hello World from GPU!\n");
}

__global__ void add(int a, int b, int *c){
	*c = a + b;
}

int main(void){
	printf("hello from cpu!\n");
	helloWorldFromGPU<<<1, 10>>>();
	//int *c;
	//int dev_c;
	//HANDLE_ERROR(hipMalloc((void**)&c, sizeof(int)));
	//add<<<1,1>>>(2, 7, c);
	//HANDLE_ERROR(hipMemcpy(&dev_c, c, sizeof(int),hipMemcpyDeviceToHost));
	//printf("dev_c=%d\n",dev_c); 
//	printf("2 + 7 = %d\n", c);
	//hipFree(c);
	hipDeviceReset();
	return 0;
}
