#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../common/book.h"

#define N 10

__global__ void add(int *a, int *b, int *c){
	int tid = blockIdx.x;
	if (tid < N){
		c[tid] = a[tid] + b[tid];
	}
}

int main(void) {
	int a[N], b[10], c[N];
	int *dev_a, *dev_b, *dev_c;

	// 在GPU上分配内存
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));

	// 在cpu上为数组‘a' 和’b'赋值
	for (int i=0; i<N; i++){
		a[i] = -i;
		b[i] = i*i;
	}

	// 将数组‘a' 和’b‘ 复制到gpu
	HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int),hipMemcpyHostToDevice));
	add<<<N, 1>>>(dev_a, dev_b, dev_c);

	// 将数组’c‘从gpu复制到cpu
	HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int),hipMemcpyDeviceToHost));

	//显示结果 
	for (int i=0; i<N; i++){
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	//释放在gpu上分配的内存
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
